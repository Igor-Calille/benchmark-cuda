#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

// Kernel para adicionar dois vetores
__global__ void add(int n, float* x, float* y, float* out) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        out[i] = x[i] + y[i];
    }
}

int main(void) {
    int N = 10000000; 
    float* x, * y, * out;

    // Alocar mem�ria unificada
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));
    hipMallocManaged(&out, N * sizeof(float));

    // Inicializar os vetores x e y no host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Configurar os blocos e threads
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;

    // Medir o tempo de execu��o do kernel
    auto start = std::chrono::high_resolution_clock::now();

    // Executar o kernel para adicionar os vetores
    add << <numBlocks, blockSize >> > (N, x, y, out);

    // Esperar a GPU finalizar antes de acessar os resultados no host
    hipDeviceSynchronize();

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float, std::milli> duration = end - start;

    // Verificar por erros (todos os valores devem ser 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++) {
        maxError = fmax(maxError, fabs(out[i] - 3.0f));
    }
    std::cout << "Max error: " << maxError << std::endl;
    std::cout << "Execution time: " << duration.count() << " ms" << std::endl;

    // Liberar mem�ria
    hipFree(x);
    hipFree(y);
    hipFree(out);

    return 0;
}
